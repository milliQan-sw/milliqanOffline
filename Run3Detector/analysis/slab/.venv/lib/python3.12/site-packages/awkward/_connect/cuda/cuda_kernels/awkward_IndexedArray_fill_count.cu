#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

template <typename T>
__global__ void
awkward_IndexedArray_fill_count(
    T* toindex,
    int64_t toindexoffset,
    int64_t length,
    int64_t base,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < length) {
      toindex[(toindexoffset + thread_id)] = (thread_id + base);
    }
  }
}
