#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

template <typename T, typename C, typename U>
__global__ void
awkward_IndexedArray_overlay_mask(
    T* toindex,
    const C* mask,
    const U* fromindex,
    int64_t length,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < length) {
      C m = mask[thread_id];
      toindex[thread_id] = (m ? -1 : fromindex[thread_id]);
    }
  }
}
