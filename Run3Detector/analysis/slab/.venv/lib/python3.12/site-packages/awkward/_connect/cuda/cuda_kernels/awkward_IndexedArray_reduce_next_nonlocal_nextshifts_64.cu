#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

// BEGIN PYTHON
// def f(grid, block, args):
//     (nextshifts, index, length, invocation_index, err_code) = args
//     scan_in_array_k = cupy.zeros(length, dtype=cupy.int64)
//     scan_in_array_nullsum = cupy.zeros(length, dtype=cupy.int64)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_IndexedArray_reduce_next_nonlocal_nextshifts_64_a", nextshifts.dtype, index.dtype]))(grid, block, (nextshifts, index, length, scan_in_array_k, scan_in_array_nullsum, invocation_index, err_code))
//     scan_in_array_k = cupy.cumsum(scan_in_array_k)
//     scan_in_array_nullsum = cupy.cumsum(scan_in_array_nullsum)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_IndexedArray_reduce_next_nonlocal_nextshifts_64_b", nextshifts.dtype, index.dtype]))(grid, block, (nextshifts, index, length, scan_in_array_k, scan_in_array_nullsum, invocation_index, err_code))
// out["awkward_IndexedArray_reduce_next_nonlocal_nextshifts_64_a", {dtype_specializations}] = None
// out["awkward_IndexedArray_reduce_next_nonlocal_nextshifts_64_b", {dtype_specializations}] = None
// END PYTHON

template <typename T, typename C>
__global__ void
awkward_IndexedArray_reduce_next_nonlocal_nextshifts_64_a(
    T* nextshifts,
    const C* index,
    int64_t length,
    int64_t* scan_in_array_k,
    int64_t* scan_in_array_nullsum,
    uint64_t invocation_code,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < length) {
      if (index[thread_id] >= 0) {
        scan_in_array_k[thread_id] = 1;
      } else {
        scan_in_array_nullsum[thread_id] = 1;
      }
    }
  }
}

template <typename T, typename C>
__global__ void
awkward_IndexedArray_reduce_next_nonlocal_nextshifts_64_b(
    T* nextshifts,
    const C* index,
    int64_t length,
    int64_t* scan_in_array_k,
    int64_t* scan_in_array_nullsum,
    uint64_t invocation_code,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < length) {
      if (index[thread_id] >= 0) {
        nextshifts[scan_in_array_k[thread_id] - 1] = scan_in_array_nullsum[thread_id];
      }
    }
  }
}
