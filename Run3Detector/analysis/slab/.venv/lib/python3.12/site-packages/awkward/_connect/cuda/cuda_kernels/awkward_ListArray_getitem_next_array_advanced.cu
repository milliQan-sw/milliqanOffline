#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

enum class LISTARRAY_GETITEM_NEXT_ARRAY_ADVANCED_ERRORS {
  STOP_LT_START,     // message: "stops[i] < starts[i]"
  STOP_GET_LEN,      // message: "stops[i] > len(content)"
  IND_OUT_OF_RANGE,  // message: "index out of range"
};

template <typename T, typename C, typename U, typename V, typename W, typename X>
__global__ void
awkward_ListArray_getitem_next_array_advanced(
    T* tocarry,
    C* toadvanced,
    const U* fromstarts,
    const V* fromstops,
    const W* fromarray,
    const X* fromadvanced,
    int64_t lenstarts,
    int64_t lencontent,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_id < lenstarts) {
      if (fromstops[thread_id] < fromstarts[thread_id]) {
        RAISE_ERROR(LISTARRAY_GETITEM_NEXT_ARRAY_ADVANCED_ERRORS::STOP_LT_START)
      }
      if ((fromstarts[thread_id] != fromstops[thread_id]) &&
          (fromstops[thread_id] > lencontent)) {
        RAISE_ERROR(LISTARRAY_GETITEM_NEXT_ARRAY_ADVANCED_ERRORS::STOP_GET_LEN)
      }
      int64_t length = fromstops[thread_id] - fromstarts[thread_id];
      int64_t regular_at = fromarray[fromadvanced[thread_id]];
      if (regular_at < 0) {
        regular_at += length;
      }
      if (!(0 <= regular_at && regular_at < length)) {
        RAISE_ERROR(LISTARRAY_GETITEM_NEXT_ARRAY_ADVANCED_ERRORS::IND_OUT_OF_RANGE)
      }
      tocarry[thread_id] = fromstarts[thread_id] + regular_at;
      toadvanced[thread_id] = thread_id;
    }
  }
}
