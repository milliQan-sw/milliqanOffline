#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

template <typename T, typename C>
__global__ void
awkward_ListOffsetArray_rpad_and_clip_axis1(
    T* toindex,
    const C* fromoffsets,
    int64_t length,
    int64_t target,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = (blockIdx.x * blockDim.x + threadIdx.x) / target;
    int64_t thready_id = (blockIdx.x * blockDim.x + threadIdx.x) % target;

    if (thread_id < length) {
      int64_t rangeval =
          (T)(fromoffsets[thread_id + 1] - fromoffsets[thread_id]);
      int64_t shorter = (target < rangeval) ? target : rangeval;

      if (thready_id < shorter) {
        toindex[thread_id * target + thready_id] =
            (T)fromoffsets[thread_id] + thready_id;
      } else if (thready_id >= shorter && thready_id < target) {
        toindex[thready_id * target + thready_id] = -1;
      }
    }
  }
}
