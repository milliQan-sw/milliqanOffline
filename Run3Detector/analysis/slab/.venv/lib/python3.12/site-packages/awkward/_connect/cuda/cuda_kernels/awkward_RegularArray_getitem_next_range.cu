#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

template <typename T>
__global__ void
awkward_RegularArray_getitem_next_range(
    T* tocarry,
    int64_t regular_start,
    int64_t step,
    int64_t length,
    int64_t size,
    int64_t nextsize,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = (blockIdx.x * blockDim.x + threadIdx.x) / nextsize;
    int64_t thready_id = (blockIdx.x * blockDim.x + threadIdx.x) % nextsize;

    if (thread_id < length) {
      tocarry[(thread_id * nextsize) + thready_id] =
          ((thread_id * size) + regular_start) + (thready_id * step);
    }
  }
}
