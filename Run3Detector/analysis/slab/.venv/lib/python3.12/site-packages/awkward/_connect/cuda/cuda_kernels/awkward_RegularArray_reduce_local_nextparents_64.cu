#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

// BEGIN PYTHON
// def f(grid, block, args):
//     (nextparents, size, length, invocation_index, err_code) = args
//     scan_in_array = cupy.ones(length * size, dtype=cupy.int64)
//     scan_in_array = cupy.cumsum(scan_in_array)
//     cuda_kernel_templates.get_function(fetch_specialization(['awkward_RegularArray_reduce_local_nextparents_64', nextparents.dtype]))(grid, block, (nextparents, size, length, scan_in_array, invocation_index, err_code))
// out["awkward_RegularArray_reduce_local_nextparents_64", {dtype_specializations}] = None
// END PYTHON

template <typename T>
__global__ void
awkward_RegularArray_reduce_local_nextparents_64(
    T* nextparents,
    int64_t size,
    int64_t length,
    int64_t* scan_in_array,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = (blockIdx.x * blockDim.x + threadIdx.x) / size;
    int64_t thready_id = (blockIdx.x * blockDim.x + threadIdx.x) % size;
    if (thread_id < length && thready_id < size) {
      nextparents[scan_in_array[thread_id * size + thready_id] - 1] = thread_id;
    }
  }
}
