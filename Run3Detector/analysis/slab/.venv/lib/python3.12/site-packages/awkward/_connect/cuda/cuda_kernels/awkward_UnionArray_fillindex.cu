#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

template <typename T, typename C>
__global__ void
awkward_UnionArray_fillindex(
    T* toindex,
    int64_t toindexoffset,
    const C* fromindex,
    int64_t length,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < length) {
      toindex[toindexoffset + thread_id] = (T)fromindex[thread_id];
    }
  }
}
