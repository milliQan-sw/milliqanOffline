#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

template <typename T>
__global__ void
awkward_UnionArray_fillindex_count(
    T* toindex,
    int64_t toindexoffset,
    int64_t length,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < length) {
      toindex[toindexoffset + thread_id] = (T)thread_id;
    }
  }
}
