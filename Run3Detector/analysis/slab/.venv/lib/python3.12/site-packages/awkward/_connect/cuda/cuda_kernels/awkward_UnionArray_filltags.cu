#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

template <typename T, typename C>
__global__ void
awkward_UnionArray_filltags(
    T* totags,
    int64_t totagsoffset,
    const C* fromtags,
    int64_t length,
    int64_t base,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < length) {
      totags[totagsoffset + thread_id] = (T)(fromtags[thread_id] + base);
    }
  }
}
