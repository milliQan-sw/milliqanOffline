#include "hip/hip_runtime.h"
// BSD 3-Clause License; see https://github.com/scikit-hep/awkward-1.0/blob/main/LICENSE

// BEGIN PYTHON
// def f(grid, block, args):
//     (lenout, tocarry, fromtags, fromindex, length, which, invocation_index, err_code) = args
//     scan_in_array = cupy.zeros(length, dtype=cupy.int64)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_UnionArray_project_a", lenout.dtype, tocarry.dtype, fromtags.dtype, fromindex.dtype]))(grid, block, (lenout, tocarry, fromtags, fromindex, length, which, scan_in_array, invocation_index, err_code))
//     scan_in_array = cupy.cumsum(scan_in_array)
//     cuda_kernel_templates.get_function(fetch_specialization(["awkward_UnionArray_project_b", lenout.dtype, tocarry.dtype, fromtags.dtype, fromindex.dtype]))(grid, block, (lenout, tocarry, fromtags, fromindex, length, which, scan_in_array, invocation_index, err_code))
// out["awkward_UnionArray_project_a", {dtype_specializations}] = None
// out["awkward_UnionArray_project_b", {dtype_specializations}] = None
// END PYTHON

template <typename T, typename C, typename U, typename V>
__global__ void
awkward_UnionArray_project_a(
    T* lenout,
    C* tocarry,
    const U* fromtags,
    const V* fromindex,
    int64_t length,
    int64_t which,
    int64_t* scan_in_array,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < length) {
      if (fromtags[thread_id] == which) {
        scan_in_array[thread_id] = 1;
      }
    }
  }
}

template <typename T, typename C, typename U, typename V>
__global__ void
awkward_UnionArray_project_b(
    T* lenout,
    C* tocarry,
    const U* fromtags,
    const V* fromindex,
    int64_t length,
    int64_t which,
    int64_t* scan_in_array,
    uint64_t invocation_index,
    uint64_t* err_code) {
  if (err_code[0] == NO_ERROR) {
    *lenout = length > 0 ? scan_in_array[length - 1] : 0;
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < length) {
      if (fromtags[thread_id] == which) {
        tocarry[scan_in_array[thread_id] - 1] = fromindex[thread_id];
      }
    }
  }
}
